#include "hip/hip_runtime.h"
#include<iostream>
#include<random>
#include<chrono>
#include <hip/hip_runtime.h>
#include""
using namespace std;
const int MAXN = 2060, MAXR = 105;
float* a;
float* ans;
float* G, *D;

float* D_d;
float* ans_d;
float* G_d;
int n = 0, t = 0;
void init()
{
    //n = 16, t = 3;
    n = 1024, t = 3;
    a = (float*)malloc(sizeof(float) * n * n);
    G = (float*)malloc(sizeof(float) * t * t);
    ans = (float*)malloc(sizeof(float) * n * n);
    D = (float*)malloc(sizeof(float) * n * n * 8);
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            a[i * n + j] = rand() % 10 + 1;
            //cout << a[i * n + j] << " ";
        }//cout << "\n";
    }
    for (int i = 0; i < t; ++i) {
        for (int j = 0; j < t; ++j) {
            G[i * t + j] = rand() % 10 + 1;
        }
    }
}
__global__ void F_2x2_3x3(float* G, float* D, float* ans_d, int n,int t)
{
    int tx = threadIdx.x, ty = blockIdx.x;
    int cnt = ((n-4)/2+1)*tx+ty;
    
    //printf("%d %d %d\n", tx , ty , cnt);

    float g[16] = {};
    float tmp[16] = {};
    float d[16] = {};

    for (int i = 0; i < 9; ++i) {
        g[i] = G[i];
    }
    for (int i = 0; i < 16; ++i) {
        d[i] = D[cnt * 16 + i];
    }

    tmp[0] = d[0] - d[8];
    tmp[1] = d[1] - d[9];
    tmp[2] = d[2] - d[10];
    tmp[3] = d[3] - d[11];

    tmp[4] = d[4] + d[8];
    tmp[5] = d[5] + d[9];
    tmp[6] = d[6] + d[10];
    tmp[7] = d[7] + d[11];

    tmp[8] = -d[4] + d[8];
    tmp[9] = -d[5] + d[9];
    tmp[10] = -d[6] + d[10];
    tmp[11] = -d[7] + d[11];

    tmp[12] = d[4] - d[12];
    tmp[13] = d[5] - d[13];
    tmp[14] = d[6] - d[14];
    tmp[15] = d[7] - d[15];

    d[0] = tmp[0] - tmp[2];
    d[4] = tmp[4] - tmp[6];
    d[8] = tmp[8] - tmp[10];
    d[12] = tmp[12] - tmp[14];

    d[1] = tmp[1] + tmp[2];
    d[5] = tmp[5] + tmp[6];
    d[9] = tmp[9] + tmp[10];
    d[13] = tmp[13] + tmp[14];

    d[2] = -tmp[1] + tmp[2];
    d[6] = -tmp[5] + tmp[6];
    d[10] = -tmp[9] + tmp[10];
    d[14] = -tmp[13] + tmp[14];

    d[3] = tmp[1] - tmp[3];
    d[7] = tmp[5] - tmp[7];
    d[11] = tmp[9] - tmp[11];
    d[15] = tmp[13] - tmp[15];


    tmp[0] = g[0];
    tmp[1] = g[1];
    tmp[2] = g[2];

    tmp[3] = (g[0] + g[3] + g[6]) / 2;
    tmp[4] = (g[1] + g[4] + g[7]) / 2;
    tmp[5] = (g[2] + g[5] + g[8]) / 2;

    tmp[6] = (g[0] - g[3] + g[6]) / 2;
    tmp[7] = (g[1] - g[4] + g[7]) / 2;
    tmp[8] = (g[2] - g[5] + g[8]) / 2;

    tmp[9] = g[6];
    tmp[10] = g[7];
    tmp[11] = g[8];

    g[0] = tmp[0];
    g[1] = (tmp[0] + tmp[1] + tmp[2]) / 2;
    g[2] = (tmp[0] - tmp[1] + tmp[2]) / 2;
    g[3] = tmp[2];

    g[4] = tmp[3];
    g[5] = (tmp[3] + tmp[4] + tmp[5]) / 2;
    g[6] = (tmp[3] - tmp[4] + tmp[5]) / 2;
    g[7] = tmp[5];

    g[8] = tmp[6];
    g[9] = (tmp[6] + tmp[7] + tmp[8]) / 2;
    g[10] = (tmp[6] - tmp[7] + tmp[8]) / 2;
    g[11] = tmp[8];

    g[12] = tmp[9];
    g[13] = (tmp[9] + tmp[10] + tmp[11]) / 2;
    g[14] = (tmp[9] - tmp[10] + tmp[11]) / 2;
    g[15] = tmp[11];

    d[0] = d[0] * g[0];
    d[1] = d[1] * g[1];
    d[2] = d[2] * g[2];
    d[3] = d[3] * g[3];
    d[4] = d[4] * g[4];
    d[5] = d[5] * g[5];
    d[6] = d[6] * g[6];
    d[7] = d[7] * g[7];
    d[8] = d[8] * g[8];
    d[9] = d[9] * g[9];
    d[10] = d[10] * g[10];
    d[11] = d[11] * g[11];
    d[12] = d[12] * g[12];
    d[13] = d[13] * g[13];
    d[14] = d[14] * g[14];
    d[15] = d[15] * g[15];

    tmp[0] = d[0] + d[4] + d[8];
    tmp[1] = d[1] + d[5] + d[9];
    tmp[2] = d[2] + d[6] + d[10];
    tmp[3] = d[3] + d[7] + d[11];

    tmp[4] = d[4] - d[8] - d[12];
    tmp[5] = d[5] - d[9] - d[13];
    tmp[6] = d[6] - d[10] - d[14];
    tmp[7] = d[7] - d[11] - d[15];

    tx *= 2; 
    ty *= 2;
    ans_d[tx*(n-t+1)+ty] = (tmp[0] + tmp[1] + tmp[2]);
    ans_d[(tx+1)*(n-t+1)+ty] = (tmp[4] + tmp[5] + tmp[6]);
    ans_d[tx*(n-t+1)+ty+1] = (tmp[1] - tmp[2] - tmp[3]);
    ans_d[(tx+1)*(n-t+1)+ty+1] = (tmp[5] - tmp[6] - tmp[7]);
}
void img2col()
{
    int cnt = -1;
    for (int i = 0; i <= n - 4; i += 2) {
        for (int j = 0; j <= n - 4; j += 2) {
            ++cnt;
            //printf("%d %d %d\n", i/2, j/2, cnt);
            D[cnt*16] = a[i * n + j]; D[cnt*16+1] = a[i * n + j + 1];
            D[cnt*16+2] = a[i * n + j + 2]; D[cnt*16+3] = a[i * n + j + 3];

            D[cnt*16+4] = a[(i + 1) * n + j]; D[cnt*16+5] = a[(i + 1) * n + j + 1];
            D[cnt*16+6] = a[(i + 1) * n + j + 2]; D[cnt*16+7] = a[(i + 1) * n + j + 3];

            D[cnt*16+8] = a[(i + 2) * n + j]; D[cnt*16+9] = a[(i + 2) * n + j + 1];
            D[cnt*16+10] = a[(i + 2) * n + j + 2]; D[cnt*16+11] = a[(i + 2) * n + j + 3];

            D[cnt*16+12] = a[(i + 3) * n + j]; D[cnt*16+13] = a[(i + 3) * n + j + 1];
            D[cnt*16+14] = a[(i + 3) * n + j + 2]; D[cnt*16+15] = a[(i + 3) * n + j + 3];
        }
    }
}
void Wiongrad()
{
    init();
    img2col();

    hipMalloc((float**)&D_d, sizeof(float) * n * n * 8);
    hipMalloc((float**)&G_d, sizeof(float) * t * t);
    hipMalloc((float**)&ans_d, sizeof(float) * n * n);

    hipMemcpy(D_d, D, sizeof(float) * n * n * 8, hipMemcpyHostToDevice);
    hipMemcpy(G_d, G, sizeof(float) * t * t, hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();

    dim3 Winograd_grid((n - 4) / 2 + 1);
    dim3 Winograd_block((n - 4) / 2 + 1);

    F_2x2_3x3 << <Winograd_grid, Winograd_block >> > (G_d, D_d, ans_d, n, t);

    auto finish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = finish - start;
    printf("%f\n", elapsed.count());

    hipMemcpy(ans, ans_d, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    hipFree(D_d);
    hipFree(G_d);
    hipFree(ans_d);

    free(a);
    free(G);
    free(D);
}

int main()
{
    ios::sync_with_stdio(false);
	cin.tie(0);

	Wiongrad();

	/*for (int i = 0; i < n - t + 1; ++i) {
		for (int j = 0; j < n - t + 1; ++j) {
			cout << ans[i * (n - t + 1) + j] << " ";
		}cout << "\n";
	}*/

	free(ans);
	return 0;
}