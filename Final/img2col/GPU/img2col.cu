#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <chrono>
#include <omp.h>
#include <immintrin.h>
#include <hip/hip_runtime.h>
#include""
using namespace std;
const int MAXN = 2100, MAXR = 105;
float* A, *I, *R;
float* ans;
int n = 0, t = 0;
int cnt = -1;
float* I_d, *R_d,*ans_d;
void init()
{
    n = 512, t = 31;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            A[i * n + j] = rand() % 10 + 1;
            //cout<<A[i*n+j]<<" ";
        }//cout<<"\n";
    }
    for (int i = 0; i < t; ++i) {
        for (int j = 0; j < t; ++j) {
            R[i * t + j] = rand() % 10 + 1;
        }
    }
}
void img2col()
{
    int col = t * t;
    int row = (n - t + 1) * (n - t + 1);

    for (int i = 0; i <= n - t; ++i) {
        for (int j = 0; j <= n - t; ++j) {
            ++cnt;
            for (int s1 = 0; s1 < t; ++s1) {
                for (int s2 = 0; s2 < t; ++s2) {
                    I[(s1 * t + s2) * row + cnt] = A[(i + s1) * n + j + s2];
                    //cout << I[cnt * col + (s1 * t + s2)] << " ";
                }
            }//cout << "\n";
        }
    }
    /*for (int i = 0; i < col; ++i) {
        for (int j = 0; j < row; ++j) {
            cout << I[i * row + j] << " ";
        }
        cout << "\n";
    }*/
}
//void GEMM()
//{
//    for (int i = 0; i <= cnt; ++i) {
//        for (int j = 0; j < t * t; ++j) {
//            ans[i] += I[i * (t * t) + j] * R[j];
//        }
//    }
//}
#define Block 512
#define tileX 16
__global__ void GEMM(float* A, float* B, float* C, int N, int M)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    tx = tx * tileX;
    float B_t;
    for (int k = 0; k < M; ++k) {
        B_t = B[k];
        for (int i = 0; i < tileX; ++i) {
            C[tx + i] += B_t * A[ k * N + (tx + i)];
        }
    }

}
int main()
{

	A = (float*)malloc(sizeof(float) * MAXN * MAXN);
	R = (float*)malloc(sizeof(float) * MAXR * MAXR);
	I = (float*)malloc(sizeof(float) * MAXN * MAXN * 100);
	ans = (float*)malloc(sizeof(float) * MAXN * MAXN);

	hipMalloc((float**)&I_d, sizeof(float) * MAXN * MAXN * 100);
	hipMalloc((float**)&R_d, sizeof(float) * MAXR * MAXR);
	hipMalloc((float**)&ans_d, sizeof(float) * MAXN * MAXN);
	init();

	auto start = std::chrono::high_resolution_clock::now();
	img2col();
	auto finish = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float> elapsed = finish - start;
	printf("%f\n", elapsed.count());

	int col = (n - t + 1) * (n - t + 1);
	hipMemcpy(I_d, I, sizeof(float) * col * t * t, hipMemcpyHostToDevice);
	hipMemcpy(R_d, R, sizeof(float) * t * t, hipMemcpyHostToDevice);


	dim3 GEMM_grid(col / Block + (bool)(col % Block));
	dim3 GEMM_block(Block / tileX);

	start = std::chrono::high_resolution_clock::now();

	GEMM << <GEMM_grid, GEMM_block >> > (I_d, R_d, ans_d, col, t * t);

	hipMemcpy(ans, ans_d, sizeof(float) * (n - t + 1) * (n - t + 1), hipMemcpyDeviceToHost);

	finish = std::chrono::high_resolution_clock::now();
	elapsed = finish - start;
	printf("%f\n", elapsed.count());

	/*for(int i=0;i<n-t+1;++i){
		for(int j=0;j<n-t+1;++j){
			cout<<ans[i*(n-t+1)+j]<<" ";
		}
		cout<<"\n";
	}*/

    hipFree(I_d);
    hipFree(R_d);
    hipFree(ans_d);

    free(A);
    free(R);
    free(I);
    free(ans);

	return 0;
}