#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include""
using namespace std;
const int MAXN = 4005, MAXR = 105;
class Complex
{
public:
    float Re;
    float Im;
    __device__ Complex operator-() {
        return Complex{ -Re,-Im };
    }
    __device__ Complex operator+(const Complex& b) {
        return Complex{ Re + b.Re,Im + b.Im };
    }
    __device__ Complex operator-(const Complex& b) {
        return Complex{ Re - b.Re,Im - b.Im };
    }
    __device__ Complex operator*(const Complex& b) {
        return Complex{ Re * b.Re - Im * b.Im,Im * b.Re + Re * b.Im };
    }
};
Complex* a, * Core;//b展开为一维之后的FFT结果
int n = 0, m = 0, t = 0, * c;
int l = 0, * r;
int limit = 1;
int blocks = 31;
//device
Complex* a_d, * Core_d;
int* r_d, * c_d;
__device__ void swap(Complex& a, Complex& b)
{
    Complex tmp = a;
    a = b;
    b = tmp;
}
__device__ void FFT(Complex* A, int type, int limit, int* r)
{
    const float PI = acos(-1.0);
    for (int i = 0; i < limit; ++i) {
        if (i < r[i]) {
            swap(A[i], A[r[i]]);
        }
    }
    for (int mid = 1; mid < limit; mid <<= 1) {
        Complex Wn{ cos(PI / mid),type * sin(PI / mid) };
        for (int R = mid << 1, j = 0; j < limit; j += R) {
            Complex w{ 1,0 };
            for (int k = 0; k < mid; ++k, w = w * Wn) {
                Complex x = A[j + k], y = w * A[j + mid + k];
                A[j + k] = x + y;
                A[j + mid + k] = x - y;
            }
        }
    }
    if (type == 1) { return; }
    for (int i = 0; i < limit; ++i) {
        A[i].Re = A[i].Re / limit;
        A[i].Im = A[i].Im / limit;
    }
}
void init()
{
    //cin>>n>>m>>t;
    n = 32, m = 32, t = 3;
    //n = 15,m = 15,t = 3;
    while (limit <= blocks + t - 2) {
        limit <<= 1; ++l;
    }
    for (int i = 0; i < limit; ++i) {
        r[i] = (r[i >> 1] >> 1) | ((i & 1) << (l - 1));
    }
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < m; ++j) {
            //cin>>a[i][j].Re;
            a[i * m + j].Re = rand() % 10 + 1;
            //cout << a[i*m+j].Re << " ";
        }//cout << "\n";
    }
    for (int i = 0; i < t; ++i) {
        for (int j = 0; j < t; ++j) {
            //cin>>b[i][j].Re;
            Core[i * limit + j].Re = rand() % 10 + 1;
        }
    }

}
__device__ void trans(Complex* t, int limit)
{
    for (int i = threadIdx.x; i < limit; i += limit) {
        for (int j = 0; j < i; ++j) {
            swap(t[i * limit + j], t[j * limit + i]);
        }
    }
}
__global__ void FFT_Core(Complex* Core, int* r, int limit)
{
    FFT(Core + threadIdx.x * limit, 1, limit, r);
    __syncthreads();
    trans(Core, limit);
    __syncthreads();
    FFT(Core + threadIdx.x * limit, 1, limit, r);
    __syncthreads();
    trans(Core, limit);
    __syncthreads();
}
__global__ void Conv_2D(Complex* a, Complex* Core, int* r, int* c, int limit, int blocks, int n, int m, int t)
{
    int i = blockIdx.x * blocks, j = blockIdx.y * blocks;
    __shared__ Complex t1[4096];
    for (int k = threadIdx.x; k < blocks + t; k += limit) {
        for (int s = 0; s < blocks + t; ++s) {
            t1[k * limit + s] = a[(k + i) * m + s + j];
            if (k + i > n || s + j > m) { t1[k * limit + s].Re = t1[k * limit + s].Im = 0; }
        }
    }
    __syncthreads();
    FFT(t1 + threadIdx.x * limit, 1, limit, r);
    __syncthreads();
    trans(t1, limit);
    __syncthreads();
    FFT(t1 + threadIdx.x * limit, 1, limit, r);
    __syncthreads();
    trans(t1, limit);
    __syncthreads();
    for (int k = threadIdx.x; k < limit; k += limit) {
        for (int s = 0; s < limit; ++s) {
            t1[k * limit + s] = t1[k * limit + s] * Core[k * limit + s];
        }
    }
    __syncthreads();
    FFT(t1 + threadIdx.x * limit, -1, limit, r);
    __syncthreads();
    trans(t1, limit);
    __syncthreads();
    FFT(t1 + threadIdx.x * limit, -1, limit, r);
    __syncthreads();
    trans(t1, limit);
    __syncthreads();

    /*if (blockIdx.x == 1 && blockIdx.y == 0 && threadIdx.x == 0) {
        for (int i = 0; i < limit; ++i) {
            for (int j = 0; j < limit; ++j) {
                printf("%.2f %.2fj ", t1[i * limit + j].Re, t1[i * limit + j].Im);
            }printf("\n");
        }
    }*/
    for (int k = threadIdx.x; k < blocks; k += limit) {
        for (int s = 0; s < blocks; ++s) {
            if (k + i >= n - t + 1 || s + j >= m - t + 1) { continue; }
            c[(k + i) * (m - t + 1) + s + j] = (int)t1[(k + t - 1) * limit + (s + t - 1)].Re;
        }
    }
}

int main()
{
    a = (Complex*)malloc(sizeof(Complex) * MAXN * MAXN);
    Core = (Complex*)malloc(sizeof(Complex) * MAXR * MAXR * 10);
    r = (int*)malloc(sizeof(int) * MAXN);
    c = (int*)malloc(sizeof(int) * MAXN * MAXN);
    memset(a, 0, sizeof(Complex) * MAXN * MAXN);
    memset(Core, 0, sizeof(Complex) * MAXR * MAXR * 10);
    memset(r, 0, sizeof(int) * MAXN);
    init();


    hipMalloc((Complex**)&a_d, sizeof(Complex) * n * m);
    hipMalloc((Complex**)&Core_d, sizeof(Complex) * limit * limit * 2);
    hipMalloc((int**)&r_d, sizeof(int) * limit);
    hipMalloc((int**)&c_d, sizeof(int) * n * m);

    hipMemcpy(a_d, a, sizeof(Complex) * n * m, hipMemcpyHostToDevice);
    hipMemcpy(Core_d, Core, sizeof(Complex) * limit * limit * 2, hipMemcpyHostToDevice);
    hipMemcpy(r_d, r, sizeof(int) * limit, hipMemcpyHostToDevice);

    dim3 FFT_grid(n / blocks + (bool)(n % blocks), m / blocks + (bool)(m % blocks));
    dim3 FFT_block(limit);

    auto start = std::chrono::high_resolution_clock::now();

    FFT_Core << <1, limit >> > (Core_d, r_d, limit);
    Conv_2D << <FFT_grid, FFT_block >> > (a_d, Core_d, r_d, c_d, limit, blocks, n, m, t);
    hipMemcpy(c, c_d, sizeof(int) * n * m, hipMemcpyDeviceToHost);

    /*for (int i = 0; i < n-t+1; ++i) {
        for (int j = 0; j < m-t+1; ++j) {
            cout << c[i * (m-t+1) + j] << " ";
        }
        cout << "\n";
    }*/
    auto finish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> elapsed = finish - start;
    printf("%f\n", elapsed.count());

    hipFree(a_d);
    hipFree(Core_d);
    hipFree(r_d);
    hipFree(c_d);

    free(a);
    free(Core);
    free(r);
    free(c);

    return 0;
}